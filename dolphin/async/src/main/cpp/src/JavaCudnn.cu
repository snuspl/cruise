/*
 * Copyright (C) 2016 Seoul National University
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *         http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#include <thrust/device_ptr.h>
#include <thrust/equal.h>
#include <thrust/reduce.h>

#include <stdexcept>
#include <utility>
#include <cfloat>
#include <cmath>
#include <iostream>

#include "JavaCudnn.h"

/*
 * Many cuDNN routines like hipdnnConvolutionForward take pointers to scaling factors (in host memory),
 * that are used to blend computed values with initial values in the destination tensor as follows:
 * dstValue = alpha[0]*computedValue + beta[0]*priorDstValue.
 * For improved performance it is advised to use beta[0] = 0.0.
 * Use a non-zero value for beta[0] only when blending with prior values stored in the output tensor is needed.
 * For further description refer to cuDNN API.
 * We are following Caffe in this matter, so it may need to be changed.
 */

bool cudnnCheck(const hipdnnStatus_t condition) {
  return (condition == HIPDNN_STATUS_SUCCESS);
}

void* deviceMalloc(const size_t size) {
  void* devPtr;
  if (hipSuccess != hipMalloc(&devPtr, size)) {
    devPtr = NULL;
  }
  return devPtr;
}

void freeCudnnHandle(hipdnnHandle_t* handle) {
  if (*handle != NULL) {
    hipdnnDestroy(*handle);
  }
  delete handle;
}

boost::thread_specific_ptr<hipdnnHandle_t> JavaCudnn::cudnnHandle(freeCudnnHandle);

hipdnnHandle_t JavaCudnn::getCudnnHandle() {
  if (!cudnnHandle.get()) {
    // allocate new cudnn handle
    hipdnnHandle_t* newHandlePtr = new hipdnnHandle_t;
    if (HIPDNN_STATUS_SUCCESS != hipdnnCreate(newHandlePtr)) {
      throw std::runtime_error("Cannot create Cudnn handle. Cudnn won't be available.");
    }
    cudnnHandle.reset(newHandlePtr);
  }
  return *cudnnHandle.get();
}

// Functions for creating descriptors.

hipdnnTensorDescriptor_t* JavaCudnn::createTensorDesc(const int n, const int c, const int h, const int w,
                                                     const int nStride, const int cStride, const int hStride, const int wStride) {
  hipdnnTensorDescriptor_t* tensorDesc = ((hipdnnTensorDescriptor_t*) std::malloc(sizeof(hipdnnTensorDescriptor_t)));
  if (!cudnnCheck(hipdnnCreateTensorDescriptor(tensorDesc)) ||
      !cudnnCheck(hipdnnSetTensor4dDescriptorEx(*tensorDesc, HIPDNN_DATA_FLOAT, n, c, h, w, nStride, cStride, hStride, wStride))) {
    return NULL;
  } else {
    return tensorDesc;
  }
}

hipdnnTensorDescriptor_t* JavaCudnn::createTensorDesc(const int n, const int c, const int h, const int w) {
  int wStride = 1;
  int hStride = w * wStride;
  int cStride = h * hStride;
  int nStride = c * cStride;
  return createTensorDesc(n, c, h, w, nStride, cStride, hStride, wStride);
}

hipdnnFilterDescriptor_t* JavaCudnn::createFilterDesc(const int k, const int c, const int h, const int w) {
  hipdnnFilterDescriptor_t* filterDesc = ((hipdnnFilterDescriptor_t*) std::malloc(sizeof(hipdnnFilterDescriptor_t)));
  if (!cudnnCheck(hipdnnCreateFilterDescriptor(filterDesc)) ||
      !cudnnCheck(hipdnnSetFilter4dDescriptor(*filterDesc, HIPDNN_DATA_FLOAT, HIPDNN_TENSOR_NCHW, k, c, h, w))) {
    return NULL;
  } else {
    return filterDesc;
  }
}

hipdnnConvolutionDescriptor_t* JavaCudnn::createConvDesc(const int padH, const int padW, const int strideH, const int strideW) {
  hipdnnConvolutionDescriptor_t* convDesc = ((hipdnnConvolutionDescriptor_t*) std::malloc(sizeof(hipdnnConvolutionDescriptor_t)));
  if (!cudnnCheck(hipdnnCreateConvolutionDescriptor(convDesc)) ||
      !cudnnCheck(hipdnnSetConvolution2dDescriptor(*convDesc, padH, padW, strideH, strideW, 1, 1, HIPDNN_CROSS_CORRELATION))) {
    return NULL;
  } else {
    return convDesc;
  }
}

hipdnnPoolingDescriptor_t* JavaCudnn::createPoolDesc(const char mode, const int h, const int w,
                                                    const int padH, const int padW, const int strideH, const int strideW) {
  hipdnnPoolingDescriptor_t* poolDesc = ((hipdnnPoolingDescriptor_t*) std::malloc (sizeof(hipdnnPoolingDescriptor_t)));
  hipdnnPoolingMode_t poolingMode;

  switch (mode) {
  case 'M':
    poolingMode = HIPDNN_POOLING_MAX;
    break;
  case 'A':
    poolingMode = HIPDNN_POOLING_AVERAGE_COUNT_INCLUDE_PADDING;
    break;
  default:
    return NULL;
  }

  if (!cudnnCheck(hipdnnCreatePoolingDescriptor(poolDesc)) ||
      !cudnnCheck(hipdnnSetPooling2dDescriptor(*poolDesc, poolingMode, HIPDNN_PROPAGATE_NAN, h, w, padH, padW, strideH, strideW))) {
    return NULL;
  } else {
    return poolDesc;
  }
}

hipdnnActivationDescriptor_t* JavaCudnn::createActivDesc(const char func) {
  hipdnnActivationDescriptor_t* activDesc = ((hipdnnActivationDescriptor_t*) std::malloc (sizeof(hipdnnActivationDescriptor_t)));
  hipdnnActivationMode_t activationMode;

  switch (func) {
  case 'S':
    activationMode = HIPDNN_ACTIVATION_SIGMOID;
    break;
  case 'R':
    activationMode = HIPDNN_ACTIVATION_RELU;
    break;
  case 'T':
    activationMode = HIPDNN_ACTIVATION_TANH;
    break;
  case 'C':
    activationMode = HIPDNN_ACTIVATION_CLIPPED_RELU;
    break;
  default:
    return NULL;
  }

  if(!cudnnCheck(hipdnnCreateActivationDescriptor(activDesc)) ||
     !cudnnCheck(hipdnnSetActivationDescriptor(*activDesc, activationMode, HIPDNN_PROPAGATE_NAN, 0.0))) {
    return NULL;
  } else {
    return activDesc;
  }
}

hipdnnLRNDescriptor_t* JavaCudnn::createLRNDesc(const int localSize, const float alpha, const float beta, const float k) {
  hipdnnLRNDescriptor_t* normDesc = ((hipdnnLRNDescriptor_t*) std::malloc (sizeof(hipdnnLRNDescriptor_t)));
  if(!cudnnCheck(hipdnnCreateLRNDescriptor(normDesc)) || !cudnnCheck(hipdnnSetLRNDescriptor(*normDesc, localSize, alpha, beta, k))) {
    return NULL;
  } else {
    return normDesc;
  }
}

// Functions for getting algorithm.

hipdnnConvolutionFwdAlgo_t* JavaCudnn::getConvForwardAlgo(const hipdnnTensorDescriptor_t* xDesc, const hipdnnFilterDescriptor_t* wDesc,
                                                         const hipdnnConvolutionDescriptor_t* convDesc, const hipdnnTensorDescriptor_t* yDesc) {
  hipdnnConvolutionFwdAlgo_t* algo = ((hipdnnConvolutionFwdAlgo_t*) std::malloc (sizeof(hipdnnConvolutionFwdAlgo_t)));
  *algo = (hipdnnConvolutionFwdAlgo_t) 0;

  if (cudnnCheck(hipdnnGetConvolutionForwardAlgorithm(getCudnnHandle(), *xDesc, *wDesc, *convDesc, *yDesc,
                                                     HIPDNN_CONVOLUTION_FWD_SPECIFY_WORKSPACE_LIMIT, CUDA_MEM_LIM, algo))) {
    return algo;
  } else {
    return NULL;
  }
}

hipdnnConvolutionBwdDataAlgo_t* JavaCudnn::getConvBackwardDataAlgo(const hipdnnFilterDescriptor_t* wDesc, const hipdnnTensorDescriptor_t* dyDesc,
                                                                  const hipdnnConvolutionDescriptor_t* convDesc, const hipdnnTensorDescriptor_t* dxDesc) {
  hipdnnConvolutionBwdDataAlgo_t* algo = ((hipdnnConvolutionBwdDataAlgo_t*) std::malloc (sizeof(hipdnnConvolutionBwdDataAlgo_t)));
  *algo = (hipdnnConvolutionBwdDataAlgo_t) 0;
  if (cudnnCheck(hipdnnGetConvolutionBackwardDataAlgorithm(getCudnnHandle(), *wDesc, *dyDesc, *convDesc, *dxDesc,
                                                          HIPDNN_CONVOLUTION_BWD_DATA_SPECIFY_WORKSPACE_LIMIT, CUDA_MEM_LIM, algo))) {
   return algo;
 } else {
  return NULL;
 }
}

hipdnnConvolutionBwdFilterAlgo_t* JavaCudnn::getConvBackwardFilterAlgo(const hipdnnTensorDescriptor_t* xDesc, const hipdnnTensorDescriptor_t* dyDesc,
                                                                      const hipdnnConvolutionDescriptor_t* convDesc, const hipdnnFilterDescriptor_t* dwDesc) {
  hipdnnConvolutionBwdFilterAlgo_t* algo = ((hipdnnConvolutionBwdFilterAlgo_t*) std::malloc (sizeof(hipdnnConvolutionBwdFilterAlgo_t)));
  *algo = (hipdnnConvolutionBwdFilterAlgo_t) 0;

  if (cudnnCheck(hipdnnGetConvolutionBackwardFilterAlgorithm(getCudnnHandle(), *xDesc, *dyDesc, *convDesc, *dwDesc,
                                                            HIPDNN_CONVOLUTION_BWD_FILTER_SPECIFY_WORKSPACE_LIMIT, CUDA_MEM_LIM, algo))) {
    return algo;
  }
  else {
   return NULL;
  }
}

// Functions for getting workspace.

void* JavaCudnn::getWorkspace(size_t workspaceSizeInBytes) {
  return std::malloc(workspaceSizeInBytes);
}

size_t JavaCudnn::getConvForwardWorkspaceSizeInBytes(const hipdnnTensorDescriptor_t* xDesc, const hipdnnFilterDescriptor_t* wDesc,
                                                     const hipdnnConvolutionDescriptor_t* convDesc, const hipdnnTensorDescriptor_t* yDesc,
                                                     const hipdnnConvolutionFwdAlgo_t* algo) {
  size_t fwdWorkspace = 0;
  if (cudnnCheck(hipdnnGetConvolutionForwardWorkspaceSize(getCudnnHandle(), *xDesc, *wDesc, *convDesc, *yDesc, *algo, &fwdWorkspace))) {
    return fwdWorkspace;
  } else {
    return 0;
  }
}

size_t JavaCudnn::getConvBackwardDataWorkspaceSizeInBytes(const hipdnnFilterDescriptor_t* wDesc, const hipdnnTensorDescriptor_t* dyDesc,
                                                          const hipdnnConvolutionDescriptor_t* convDesc, const hipdnnTensorDescriptor_t* dxDesc,
                                                          const hipdnnConvolutionBwdDataAlgo_t* algo) {
  size_t bwdDataWorkspace = 0;
  if (cudnnCheck(hipdnnGetConvolutionBackwardDataWorkspaceSize(getCudnnHandle(), *wDesc, *dyDesc, *convDesc, *dxDesc, *algo, &bwdDataWorkspace))) {
    return bwdDataWorkspace;
  } else {
    return 0;
  }
}

size_t JavaCudnn::getConvBackwardFilterWorkspaceSizeInBytes(const hipdnnTensorDescriptor_t* xDesc, const hipdnnTensorDescriptor_t* dyDesc,
                                                            const hipdnnConvolutionDescriptor_t* convDesc, const hipdnnFilterDescriptor_t* dwDesc,
                                                            const hipdnnConvolutionBwdFilterAlgo_t* algo) {
  size_t bwdFilterWorkspace = 0;
  if (cudnnCheck(hipdnnGetConvolutionBackwardFilterWorkspaceSize(getCudnnHandle(), *xDesc, *dyDesc, *convDesc, *dwDesc, *algo, &bwdFilterWorkspace))) {
    return bwdFilterWorkspace;
  } else {
    return 0;
  }
}

// FeedForward, BackPropagate, generateParameterGradient functions.

bool JavaCudnn::convFeedForward(const hipdnnTensorDescriptor_t* xDesc, const void* x,
                                const hipdnnFilterDescriptor_t* wDesc, const void* w,
                                const hipdnnTensorDescriptor_t* bDesc, const void* b,
                                const hipdnnConvolutionDescriptor_t* convDesc, const hipdnnConvolutionFwdAlgo_t* algo,
                                void* workspace, size_t workspaceSizeInBytes,
                                const hipdnnTensorDescriptor_t* yDesc, void* y) {
  if (cudnnCheck(hipdnnConvolutionForward(getCudnnHandle(), &CUDA_ONE ,*xDesc, x, *wDesc, w, *convDesc, *algo,
                                         workspace, workspaceSizeInBytes, &CUDA_ZERO, *yDesc, y))) {
    return cudnnCheck(hipdnnAddTensor(getCudnnHandle(), &CUDA_ONE, *bDesc, b, &CUDA_ONE, *yDesc, y));
  } else {
    return false;
  }
}

bool JavaCudnn::convBackPropagate(const hipdnnFilterDescriptor_t* wDesc, const void* w,
                                  const hipdnnTensorDescriptor_t* dyDesc, const void* dy,
                                  const hipdnnConvolutionDescriptor_t* convDesc, const hipdnnConvolutionBwdDataAlgo_t* algo,
                                  void* workspace, size_t workspaceSizeInBytes,
                                  const hipdnnTensorDescriptor_t* dxDesc, void* dx) {
  return cudnnCheck(hipdnnConvolutionBackwardData(getCudnnHandle(), &CUDA_ONE, *wDesc, w, *dyDesc, dy, *convDesc, *algo,
                                                 workspace, workspaceSizeInBytes, &CUDA_ZERO, *dxDesc, dx));
}

bool JavaCudnn::convGenWeightGradient(const hipdnnTensorDescriptor_t* xDesc, const void* x,
                                      const hipdnnTensorDescriptor_t* dyDesc, const void* dy,
                                      const hipdnnConvolutionDescriptor_t* convDesc, const hipdnnConvolutionBwdFilterAlgo_t* algo,
                                      void* workspace, size_t workspaceSizeInBytes,
                                      const hipdnnFilterDescriptor_t* dwDesc, void* dw) {
  return cudnnCheck(hipdnnConvolutionBackwardFilter(getCudnnHandle(), &CUDA_ONE, *xDesc, x, *dyDesc, dy, *convDesc, *algo,
                                                   workspace, workspaceSizeInBytes, &CUDA_ONE, *dwDesc, dw));
}

bool JavaCudnn::convGenBiasGradient(const hipdnnTensorDescriptor_t* dyDesc, const void* dy, const hipdnnTensorDescriptor_t* dbDesc, void* db) {
  return cudnnCheck(hipdnnConvolutionBackwardBias(getCudnnHandle(), &CUDA_ONE, *dyDesc, dy, &CUDA_ONE, *dbDesc, db));
}

bool JavaCudnn::poolFeedForward(const hipdnnPoolingDescriptor_t* poolDesc,
                                const hipdnnTensorDescriptor_t* xDesc, const void* x,
                                const hipdnnTensorDescriptor_t* yDesc, void* y) {
  return cudnnCheck(hipdnnPoolingForward(getCudnnHandle(), *poolDesc, &CUDA_ONE, *xDesc, x, &CUDA_ZERO, *yDesc, y));
}

bool JavaCudnn::poolBackPropagate(const hipdnnPoolingDescriptor_t* poolDesc,
                                  const hipdnnTensorDescriptor_t* yDesc, const void* y,
                                  const hipdnnTensorDescriptor_t* dyDesc, const void* dy,
                                  const hipdnnTensorDescriptor_t* xDesc, const void* x,
                                  const hipdnnTensorDescriptor_t* dxDesc, void* dx) {
  return cudnnCheck(hipdnnPoolingBackward(getCudnnHandle(), *poolDesc, &CUDA_ONE, *yDesc, y, *dyDesc, dy, *xDesc, x, &CUDA_ZERO, *dxDesc, dx));
}

bool JavaCudnn::activFeedForward(const hipdnnActivationDescriptor_t* activDesc,
                                 const hipdnnTensorDescriptor_t* srcDesc, const void* src,
                                 const hipdnnTensorDescriptor_t* destDesc, void* dest) {
  return cudnnCheck(hipdnnActivationForward(getCudnnHandle(), *activDesc, &CUDA_ONE, *srcDesc, src, &CUDA_ZERO, *destDesc, dest));
}

bool JavaCudnn::activBackPropagate(const hipdnnActivationDescriptor_t* activDesc,
                                   const hipdnnTensorDescriptor_t* srcDesc, const void* src,
                                   const hipdnnTensorDescriptor_t* srcDiffDesc, const void* srcDiff,
                                   const hipdnnTensorDescriptor_t* destDesc, const void* dest,
                                   const hipdnnTensorDescriptor_t* destDiffDesc, void* destDiff ) {
  return cudnnCheck(hipdnnActivationBackward(getCudnnHandle(), *activDesc, &CUDA_ONE, *srcDesc, src,
                                            *srcDiffDesc, srcDiff, *destDesc, dest, &CUDA_ZERO, *destDiffDesc, destDiff));
}

bool JavaCudnn::activWithLossFeedForward (const hipdnnTensorDescriptor_t* xDesc, const void* x,
                                          const hipdnnTensorDescriptor_t* yDesc, void* y) {
  return cudnnCheck(hipdnnSoftmaxForward(getCudnnHandle(), HIPDNN_SOFTMAX_ACCURATE, HIPDNN_SOFTMAX_MODE_CHANNEL, &CUDA_ONE,
                                        *xDesc, x, &CUDA_ZERO, *yDesc, y));
}
bool JavaCudnn::activWithLossBackPropagate (const hipdnnTensorDescriptor_t* yDesc, const void* y,
                                            const hipdnnTensorDescriptor_t* dyDesc, const void* dy,
                                            const hipdnnTensorDescriptor_t* dxDesc, void* dx) {
  return cudnnCheck(hipdnnSoftmaxBackward(getCudnnHandle(), HIPDNN_SOFTMAX_ACCURATE, HIPDNN_SOFTMAX_MODE_CHANNEL, &CUDA_ONE,
                                         *yDesc, y, *dyDesc, dy, &CUDA_ZERO, *dxDesc, dx));
}

bool JavaCudnn::lrnFeedForward(const hipdnnLRNDescriptor_t* normDesc, const hipdnnTensorDescriptor_t* xDesc, const void* x,
                               const hipdnnTensorDescriptor_t* yDesc, void* y) {
 return cudnnCheck(hipdnnLRNCrossChannelForward(getCudnnHandle(), *normDesc, HIPDNN_LRN_CROSS_CHANNEL, &CUDA_ONE, *xDesc, x, &CUDA_ZERO, *yDesc, y));
}

bool JavaCudnn::lrnBackPropagate(const hipdnnLRNDescriptor_t* normDesc,
                                 const hipdnnTensorDescriptor_t* yDesc, const void* y,
                                 const hipdnnTensorDescriptor_t* dyDesc, const void* dy,
                                 const hipdnnTensorDescriptor_t* xDesc, const void* x,
                                 const hipdnnTensorDescriptor_t* dxDesc, void* dx) {
 return cudnnCheck(hipdnnLRNCrossChannelBackward(getCudnnHandle(), *normDesc, HIPDNN_LRN_CROSS_CHANNEL, &CUDA_ONE,
                                                *yDesc, y, *dyDesc, dy, *xDesc, x, &CUDA_ZERO, *dxDesc, dx));
}
