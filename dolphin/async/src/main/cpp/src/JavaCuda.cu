#include "hip/hip_runtime.h"
/*
 * Copyright (C) 2016 Seoul National University
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *         http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#include <thrust/device_ptr.h>
#include <thrust/equal.h>
#include <thrust/reduce.h>

#include <stdexcept>
#include <utility>
#include <cfloat>
#include <cmath>

#include "JavaCuda.h"

#define CUDA_KERNEL_LOOP(i, n) \
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; \
       i < (n); \
       i += blockDim.x * gridDim.x)

struct float_compare {
  float tolerance;
  float_compare(const float t) : tolerance(t) {
  }
  __device__ bool operator()(const float x, const float y) const {
    return abs(x - y) < tolerance;
  }
};

std::pair<hipblasOperation_t, bool> getCublasOperation(const char c) {
  switch (c) {
  case 'N':
  case 'n':
    return std::make_pair(HIPBLAS_OP_N, true);
  case 'T':
  case 't':
    return std::make_pair(HIPBLAS_OP_T, true);
  case 'C':
  case 'c':
    return std::make_pair(HIPBLAS_OP_C, true);
  default:
    return std::make_pair(HIPBLAS_OP_N, false);
  }
}

void freeCublasHandle(hipblasHandle_t* handle) {
  if (*handle != NULL) {
    hipblasDestroy(*handle);
  }
  delete handle;
}

boost::thread_specific_ptr<hipblasHandle_t> JavaCuda::cublasHandle(freeCublasHandle);

/*
 * Get cuBLAS handle which is required for every functions in cuBLAS.
 * All threads have different cuBLAS handle pointer by using boost thread specific pointer.
 * When a thread requests for cuBLAS handle, it checks whether thread specific pointer for cuBLAS handle is set.
 * If not, new cuBLAS handle is created.
 * Thread specific pointer is destroyed by freeCublasHandle().
 * This destroying is automatically done by boost when the thread is killed.
 */
hipblasHandle_t JavaCuda::getCublasHandle() {
  if (!cublasHandle.get()) {
    // allocate new cublas handle
    hipblasHandle_t* newHandlePtr = new hipblasHandle_t;
    if (HIPBLAS_STATUS_SUCCESS != hipblasCreate(newHandlePtr)) {
      throw std::runtime_error("Cannot create Cublas handle. Cublas won't be available.");
    }
    cublasHandle.reset(newHandlePtr);
  }
  return *cublasHandle.get();
}

void* JavaCuda::cudaDeviceMalloc(size_t size) {
  void* devPtr;
  if (hipSuccess != hipMalloc(&devPtr, size)) {
    devPtr = NULL;
  }
  return devPtr;
}

bool JavaCuda::cudaDeviceFree(void* devPtr) {
  return hipSuccess == hipFree(devPtr);
}

bool JavaCuda::d2hMemcpy(void* dst, const void* src, const int n) {
  return hipSuccess == hipMemcpy(dst, src, n, hipMemcpyDeviceToHost);
}

bool JavaCuda::h2dMemcpy(void* dst, const void* src, const int n) {
  return hipSuccess == hipMemcpy(dst, src, n, hipMemcpyHostToDevice);
}

bool JavaCuda::d2dMemcpy(void* dst, const void* src, const int n) {
  return hipSuccess == hipMemcpy(dst, src, n, hipMemcpyDeviceToDevice);
}

bool JavaCuda::d2dMemcpy2D(void* dst, const int dpitch, const void* src, const int spitch, const int length, const int n) {
  return hipSuccess == hipMemcpy2D(dst, dpitch, src, spitch, length, n, hipMemcpyDeviceToDevice);
}

__global__ void set_kernel(const int n, const float a, float* y) {
  CUDA_KERNEL_LOOP(i, n) {
    y[i] = a;
  }
}

bool JavaCuda::set(float* y, const float a, const int n) {
  if (a == 0) {
    return hipSuccess == hipMemset(y, 0, sizeof(float) * n);
  }

  set_kernel<<<GET_BLOCKS(n), CUDA_NUM_THREADS>>>(n, a, y);
  return true;
}

bool JavaCuda::copy(const int n, const float* x, int incx, float* y, int incy) {
  return HIPBLAS_STATUS_SUCCESS == hipblasScopy(getCublasHandle(), n, x, incx, y, incy);
}

bool JavaCuda::compare(const int n, const float* x, const float* y, const float tolerance) {
  thrust::device_ptr<float> device_ptr_x(const_cast<float*>(x));
  thrust::device_ptr<float> device_ptr_y(const_cast<float*>(y));
  return thrust::equal(device_ptr_x, device_ptr_x + n, device_ptr_y, float_compare(tolerance));
}

float JavaCuda::sum(const int n, const float* x) {
  thrust::device_ptr<float> device_ptr(const_cast<float*>(x));
  return thrust::reduce(device_ptr, device_ptr + n, 0.0f, thrust::plus<float>());
}

float JavaCuda::max(const int n, const float* x) {
  thrust::device_ptr<float> device_ptr(const_cast<float*>(x));
  return thrust::reduce(device_ptr, device_ptr + n, FLT_MIN, thrust::maximum<float>());
}

float JavaCuda::min(const int n, const float* x) {
  thrust::device_ptr<float> device_ptr(const_cast<float*>(x));
  return thrust::reduce(device_ptr, device_ptr + n, FLT_MAX, thrust::minimum<float>());
}

__global__ void column_max_kernel(const int m, const int n, const float* x, float* y) {
  CUDA_KERNEL_LOOP(i, n) {
    float max = FLT_MIN;
    for (int j = m * i; j < m * (i + 1); ++j) {
      max = max < x[j] ? x[j] : max;
    }
    y[i] = max;
  }
}

bool JavaCuda::columnMax(const int m, const int n, const float* x, float* y) {
  column_max_kernel<<<GET_BLOCKS(n), CUDA_NUM_THREADS>>>(m, n, x, y);
  return true;
}

__global__ void row_max_kernel(const int m, const int n, const float* x, float* y) {
  CUDA_KERNEL_LOOP(i, m) {
    float max = FLT_MIN;
    for (int j = i; j < m * n; j += m) {
      max = max < x[j] ? x[j] : max;
    }
    y[i] = max;
  }
}

bool JavaCuda::rowMax(const int m, const int n, const float* x, float* y) {
  row_max_kernel<<<GET_BLOCKS(m), CUDA_NUM_THREADS>>>(m, n, x, y);
  return true;
}

__global__ void column_min_kernel(const int m, const int n, const float* x, float* y) {
  CUDA_KERNEL_LOOP(i, n) {
    float min = FLT_MAX;
    for (int j = m * i; j < m * (i + 1); ++j) {
      min = min > x[j] ? x[j] : min;
    }
    y[i] = min;
  }
}

bool JavaCuda::columnMin(const int m, const int n, const float* x, float* y) {
  column_min_kernel<<<GET_BLOCKS(n), CUDA_NUM_THREADS>>>(m, n, x, y);
  return true;
}

__global__ void row_min_kernel(const int m, const int n, const float* x, float* y) {
  CUDA_KERNEL_LOOP(i, m) {
    float min = FLT_MAX;
    for (int j = i; j < m * n; j += m) {
      min = min > x[j] ? x[j] : min;
    }
    y[i] = min;
  }
}

bool JavaCuda::rowMin(const int m, const int n, const float* x, float* y) {
  row_min_kernel<<<GET_BLOCKS(m), CUDA_NUM_THREADS>>>(m, n, x, y);
  return true;
}

__global__ void add_scalar_kernel(const int n, const float a, const float* x, float* y) {
  CUDA_KERNEL_LOOP(i, n) {
    y[i] = x[i] + a;
  }
}

bool JavaCuda::addScalar(const int n, const float a, const float* x, float* y) {
  add_scalar_kernel<<<GET_BLOCKS(n), CUDA_NUM_THREADS>>>(n, a, x, y);
  return true;
}

bool JavaCuda::mulScalar(const int n, const float a, const float* x, float* y) {
  if (x != y) {
    if (!JavaCuda::d2dMemcpy(y, x, n)) {
      return false;
    }
  }
  return HIPBLAS_STATUS_SUCCESS == hipblasSscal(getCublasHandle(), n, &a, y, 1);
}

__global__ void rsub_scalar_kernel(const int n, const float a, const float* x, float* y) {
  CUDA_KERNEL_LOOP(i, n) {
    y[i] = a - x[i];
  }
}

bool JavaCuda::rsubScalar(const int n, const float a, const float* x, float* y) {
  rsub_scalar_kernel<<<GET_BLOCKS(n), CUDA_NUM_THREADS>>>(n, a, x, y);
  return true;
}

__global__ void rdiv_scalar_kernel(const int n, const float a, const float* x, float* y) {
  CUDA_KERNEL_LOOP(i, n) {
    y[i] = a / x[i];
  }
}

bool JavaCuda::rdivScalar(const int n, const float a, const float* x, float* y) {
  rdiv_scalar_kernel<<<GET_BLOCKS(n), CUDA_NUM_THREADS>>>(n, a, x, y);
  return true;
}

__global__ void mul_column_vector_kernel(const int m, const int n, const float* v, float* x, float* y) {
  CUDA_KERNEL_LOOP(i, m * n) {
    y[i] = x[i] * v[i % m];
  }
}

bool JavaCuda::mulColumnVector(const int m, const int n, const float* v, float* x, float* y) {
  mul_column_vector_kernel<<<GET_BLOCKS(m * n), CUDA_NUM_THREADS>>>(m, n, v, x, y);
  return true;
}

__global__ void mul_row_vector_kernel(const int m, const int n, const float* v, float* x, float* y) {
  CUDA_KERNEL_LOOP(i, m * n) {
    y[i] = x[i] * v[i / m];
  }
}

bool JavaCuda::mulRowVector(const int m, const int n, const float* v, float* x, float* y) {
  mul_row_vector_kernel<<<GET_BLOCKS(m * n), CUDA_NUM_THREADS>>>(m, n, v, x, y);
  return true;
}

__global__ void div_column_vector_kernel(const int m, const int n, const float* v, float* x, float* y) {
  CUDA_KERNEL_LOOP(i, m * n) {
    y[i] = x[i] / v[i % m];
  }
}

bool JavaCuda::divColumnVector(const int m, const int n, const float* v, float* x, float* y) {
  div_column_vector_kernel<<<GET_BLOCKS(m * n), CUDA_NUM_THREADS>>>(m, n, v, x, y);
  return true;
}

__global__ void div_row_vector_kernel(const int m, const int n, const float* v, float* x, float* y) {
  CUDA_KERNEL_LOOP(i, m * n) {
    y[i] = x[i] / v[i / m];
  }
}

bool JavaCuda::divRowVector(const int m, const int n, const float* v, float* x, float* y) {
  div_row_vector_kernel<<<GET_BLOCKS(m * n), CUDA_NUM_THREADS>>>(m, n, v, x, y);
  return true;
}

__global__ void add_kernel(const int n, const float* a, const float* b, float* y) {
  CUDA_KERNEL_LOOP(i, n) {
    y[i] = a[i] + b[i];
  }
}

bool JavaCuda::add(const int n, const float* a, const float* b, float* y) {
  add_kernel<<<GET_BLOCKS(n), CUDA_NUM_THREADS>>>(n, a, b, y);
  return true;
}

__global__ void sub_kernel(const int n, const float* a, const float* b, float* y) {
  CUDA_KERNEL_LOOP(i, n) {
    y[i] = a[i] - b[i];
  }
}

bool JavaCuda::sub(const int n, const float* a, const float* b, float* y) {
  sub_kernel<<<GET_BLOCKS(n), CUDA_NUM_THREADS>>>(n, a, b, y);
  return true;
}

__global__ void mul_kernel(const int n, const float* a, const float* b, float* y) {
  CUDA_KERNEL_LOOP(i, n) {
    y[i] = a[i] * b[i];
  }
}

bool JavaCuda::mul(const int n, const float* a, const float* b, float* y) {
  mul_kernel<<<GET_BLOCKS(n), CUDA_NUM_THREADS>>>(n, a, b, y);
  return true;
}

__global__ void div_kernel(const int n, const float* a, const float* b, float* y) {
  CUDA_KERNEL_LOOP(i, n) {
    y[i] = a[i] / b[i];
  }
}

bool JavaCuda::div(const int n, const float* a, const float* b, float* y) {
  div_kernel<<<GET_BLOCKS(n), CUDA_NUM_THREADS>>>(n, a, b, y);
  return true;
}

bool JavaCuda::geam(const char transa, const char transb, const int m, const int n,
                    const float alpha, const float* a, const int lda,
                    const float beta, const float* b, const int ldb,
                    float* c, const int ldc) {
  std::pair<hipblasOperation_t, bool> opa = getCublasOperation(transa);
  std::pair<hipblasOperation_t, bool> opb = getCublasOperation(transb);
  if (!opa.second || !opb.second) {
    return false;
  }
  return HIPBLAS_STATUS_SUCCESS == hipblasSgeam(getCublasHandle(), opa.first, opb.first, m, n, &alpha, a, lda, &beta, b, ldb, c, ldc);
}

bool JavaCuda::gemv(const char trans, const int m, const int n,
                    const float alpha, const float* a, const int lda,
                    const float* x, const int incx,
                    const float beta, float* y, const int incy) {
  std::pair<hipblasOperation_t, bool> op = getCublasOperation(trans);
  if (!op.second) {
    return false;
  }
  return HIPBLAS_STATUS_SUCCESS == hipblasSgemv(getCublasHandle(), op.first, m, n, &alpha, a, lda, x, incx, &beta, y, incy);
}

bool JavaCuda::gemm(const char transa, const char transb, const int m, const int n, const int k,
                    const float alpha, const float* a, const int lda,
                    const float* b, const int ldb,
                    const float beta, float* c, const int ldc) {
  std::pair<hipblasOperation_t, bool> opa = getCublasOperation(transa);
  std::pair<hipblasOperation_t, bool> opb = getCublasOperation(transb);
  if (!opa.second || !opb.second) {
    return false;
  }
  return HIPBLAS_STATUS_SUCCESS == hipblasSgemm(getCublasHandle(), opa.first, opb.first, m, n, k, &alpha, a, lda, b, ldb, &beta, c, ldc);
}
